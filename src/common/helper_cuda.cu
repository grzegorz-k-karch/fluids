#include "hip/hip_runtime_api.h"

//==============================================================================
void myCudaCall(hipError_t hipError_t, int line, char *file)
{
  if (hipError_t) {
    fprintf(stderr, "[CUDA] Error %d in file:\n%s: \nline %d: %s\n",
	    hipError_t, file, line, hipGetErrorString(hipGetLastError()));
    exit(1);
  }
}
//==============================================================================
