#include "hip/hip_runtime.h"
#include "compute.h"
#include "hip/hip_runtime_api.h"
#include <iostream>
#include <cuda_gl_interop.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>

//==============================================================================
void Compute::Init(dataInfo_t* dataInfo)
{
  cudaGLSetGLDevice(0);
  DataInfo = dataInfo;

  NumCellFaces[0] = 
    (DataInfo->resolution[0]+1)*
    (DataInfo->resolution[1])*
    (DataInfo->resolution[2]);
  NumCellFaces[1] = 
    (DataInfo->resolution[0])*
    (DataInfo->resolution[1]+1)*
    (DataInfo->resolution[2]);
  NumCellFaces[2] = 
    (DataInfo->resolution[0])*
    (DataInfo->resolution[1])*
    (DataInfo->resolution[2]+1);

  InitData();
  InitTextures();
  InitSymbols();

  Res[0][0] = DataInfo->resolution[0]+1;
  Res[0][1] = DataInfo->resolution[1];
  Res[0][2] = DataInfo->resolution[2];
  Res[1][0] = DataInfo->resolution[0];
  Res[1][1] = DataInfo->resolution[1]+1;
  Res[1][2] = DataInfo->resolution[2];
  Res[2][0] = DataInfo->resolution[0];
  Res[2][1] = DataInfo->resolution[1];
  Res[2][2] = DataInfo->resolution[2]+1;
  Res[3][0] = DataInfo->resolution[0];
  Res[3][1] = DataInfo->resolution[1];
  Res[3][2] = DataInfo->resolution[2];

  VolumeSize.x = DataInfo->resolution[0];
  VolumeSize.y = DataInfo->resolution[1];
  VolumeSize.z = DataInfo->resolution[2];
}
//==============================================================================
void Compute::InitData()
{
  int numCells = 
    (DataInfo->resolution[0])*
    (DataInfo->resolution[1])*
    (DataInfo->resolution[2]);
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
  int res[3] = {DataInfo->resolution[0],
		DataInfo->resolution[1],
		DataInfo->resolution[2]};
  hipExtent volumeSize;

  // Dye
  myCudaCall(hipMalloc((void**)&Dye, numCells*sizeof(float)), 
	     __LINE__, __FILE__);
  volumeSize = make_hipExtent(res[0], res[1], res[2]);
  myCudaCall(hipMalloc3DArray(&ca_Dye, &channelDesc, volumeSize),
	     __LINE__, __FILE__);
  // Velocity X component
  myCudaCall(hipMalloc((void**)&VelocityX, NumCellFaces[0]*sizeof(float)), 
	     __LINE__, __FILE__);
  volumeSize = make_hipExtent(res[0]+1, res[1], res[2]);
  myCudaCall(hipMalloc3DArray(&ca_VelocityX, &channelDesc, volumeSize),
	     __LINE__, __FILE__);
  // Velocity Y component
  myCudaCall(hipMalloc((void**)&VelocityY, NumCellFaces[1]*sizeof(float)), 
	     __LINE__, __FILE__);
  volumeSize = make_hipExtent(res[0], res[1]+1, res[2]);
  myCudaCall(hipMalloc3DArray(&ca_VelocityY, &channelDesc, volumeSize),
	     __LINE__, __FILE__);
  // Velocity Z component
  myCudaCall(hipMalloc((void**)&VelocityZ, NumCellFaces[2]*sizeof(float)), 
	     __LINE__, __FILE__);
  volumeSize = make_hipExtent(res[0], res[1], res[2]+1);
  myCudaCall(hipMalloc3DArray(&ca_VelocityZ, &channelDesc, volumeSize),
	     __LINE__, __FILE__);
  // Divergence
  myCudaCall(hipMalloc((void**)&NegDivergence, numCells*sizeof(float)), 
	     __LINE__, __FILE__);
  // Pressure
  myCudaCall(hipMalloc((void**)&Pressure, numCells*sizeof(float)), 
	     __LINE__, __FILE__);
  // Boundary conditions
  myCudaCall(hipMallocArray(&ca_BCLeft, &channelDesc, res[1], res[2]),
	     __LINE__, __FILE__);
  myCudaCall(hipMallocArray(&ca_BCRight, &channelDesc, res[1], res[2]),
	     __LINE__, __FILE__);
  myCudaCall(hipMallocArray(&ca_BCBottom, &channelDesc, res[0], res[2]),
	     __LINE__, __FILE__);
  myCudaCall(hipMallocArray(&ca_BCTop, &channelDesc, res[0], res[2]),
	     __LINE__, __FILE__);
  myCudaCall(hipMallocArray(&ca_BCBack, &channelDesc, res[0], res[1]),
	     __LINE__, __FILE__);
  myCudaCall(hipMallocArray(&ca_BCFront, &channelDesc, res[0], res[1]),
	     __LINE__, __FILE__);
}
//==============================================================================
void Compute::InitDye()
{
  InitDye_kernel();

  hipGraphicsMapResources(1, &VolumeResource);
  hipGraphicsSubResourceGetMappedArray(&ca_Dye, VolumeResource, 0, 0);

  UpdateCudaArray(ca_Dye, DataInfo->resolution, Dye);

  hipGraphicsUnmapResources(1, &VolumeResource);
}
//==============================================================================
void Compute::InitVelocity()
{
  InitVelocity_kernel();

  UpdateCudaArray(ca_VelocityX, Res[0], VelocityX);
  UpdateCudaArray(ca_VelocityY, Res[1], VelocityY);
  UpdateCudaArray(ca_VelocityZ, Res[2], VelocityZ);
}
//==============================================================================
void Compute::RegisterVolumeTexture(GLuint volume)
{
  hipGraphicsGLRegisterImage(&VolumeResource, volume, GL_TEXTURE_3D, 
			      hipGraphicsRegisterFlagsNone);
}
//==============================================================================
void Compute::UnregisterVolumeTexture()
{
  hipGraphicsUnregisterResource(VolumeResource);
}
//==============================================================================
void Compute::AdvectDye()
{
  AdvectDye_kernel();

  hipGraphicsMapResources(1, &VolumeResource);
  cudaGraphicsResourceSetMapFlags(VolumeResource, 
				  cudaGraphicsMapFlagsWriteDiscard);
  hipGraphicsSubResourceGetMappedArray(&ca_Dye, VolumeResource, 0, 0);
  UpdateCudaArray(ca_Dye, DataInfo->resolution, Dye);
  hipGraphicsUnmapResources(1, &VolumeResource);
}
//==============================================================================
void Compute::AdvectVelocity()
{
  AdvectVelocity_kernel();

  UpdateCudaArray(ca_VelocityX, Res[0], VelocityX);
  UpdateCudaArray(ca_VelocityY, Res[1], VelocityY);
  UpdateCudaArray(ca_VelocityZ, Res[2], VelocityZ);
}
//==============================================================================
void Compute::SetBoundaryConditions()
{
  SetBoundaryConditions_kernel();

  float *velobc;
  int count;

  count = Res[3][1]*Res[3][2];
  velobc = new float[count];

  for (int k = 0; k < Res[3][2]; k++) {
    for (int j = 0; j < Res[3][1]; j++) {

      int idx = j + k*Res[3][1];
      if (j > Res[3][1]/4 && j < Res[3][1]*3/4 &&
  	  k > Res[3][2]/4 && k < Res[3][2]*3/4 ) {
  	velobc[idx] = 1.0f;
      }
      else {
  	velobc[idx] = 0.0f;
      }
    }
  }  
  myCudaCall(hipMemcpyToArray(ca_BCLeft, 0, 0, velobc, count, 
			       hipMemcpyHostToDevice), __LINE__, __FILE__);

  for (int k = 0; k < Res[3][2]; k++) {
    for (int j = 0; j < Res[3][1]; j++) {

      int idx = j + k*Res[3][1];
      velobc[idx] = 0.0f;     
    }
  }  
  myCudaCall(hipMemcpyToArray(ca_BCRight, 0, 0, velobc, count,
			       hipMemcpyHostToDevice), __LINE__, __FILE__);
  delete [] velobc;

  count = Res[3][0]*Res[3][2];
  velobc = new float[count];

  for (int k = 0; k < Res[3][2]; k++) {
    for (int i = 0; i < Res[3][0]; i++) {

      int idx = i + k*Res[3][0];
      velobc[idx] = 0.0f;
    }
  }  
  myCudaCall(hipMemcpyToArray(ca_BCBottom, 0, 0, velobc, count,
			       hipMemcpyHostToDevice), __LINE__, __FILE__);
  myCudaCall(hipMemcpyToArray(ca_BCTop, 0, 0, velobc, count,
			       hipMemcpyHostToDevice), __LINE__, __FILE__);
  delete [] velobc;

  count = Res[3][0]*Res[3][1];
  velobc = new float[count];

  for (int j = 0; j < Res[3][1]; j++) {
    for (int i = 0; i < Res[3][0]; i++) {

      int idx = i + j*Res[3][0];
      velobc[idx] = 0.0f;
    }
  }  
  myCudaCall(hipMemcpyToArray(ca_BCBottom, 0, 0, velobc, count,
			       hipMemcpyHostToDevice), __LINE__, __FILE__);
  myCudaCall(hipMemcpyToArray(ca_BCTop, 0, 0, velobc, count,
			       hipMemcpyHostToDevice), __LINE__, __FILE__);
  delete [] velobc;
}
//==============================================================================
void Compute::Update()
{
  SetTimestep(ComputeTimestep());
  //  SetBoundaryConditions();
  ComputeNegDivergence();
  // //Projection();
  // PressureUpdate();
  AdvectDye();
  AdvectVelocity();
}
//==============================================================================
void Compute::UpdateCudaArray(hipArray* ca, int res[3], float* src)
{
  hipMemcpy3DParms copyParams[1] = {0};  
  hipExtent volumeSize = make_hipExtent(res[0], res[1], res[2]);
  copyParams[0].srcPtr = make_hipPitchedPtr(src, res[0]*sizeof(float), 
					     res[0], res[1]);
  copyParams[0].dstArray = ca;
  copyParams[0].extent = volumeSize;
  copyParams[0].kind = hipMemcpyDeviceToDevice;

  hipMemcpy3D(copyParams);  
}
//==============================================================================
void Compute::CopyCudaArray(hipArray* ca, int res[3], float* dst)
{
  hipMemcpy3DParms copyParams[1] = {0};  
  hipExtent volumeSize = make_hipExtent(res[0], res[1], res[2]);
  copyParams[0].dstPtr = make_hipPitchedPtr(dst, res[0]*sizeof(float), 
					     res[0], res[1]);
  copyParams[0].srcArray = ca;
  copyParams[0].extent = volumeSize;
  copyParams[0].kind = hipMemcpyDeviceToDevice;

  hipMemcpy3D(copyParams);  
}
//==============================================================================
void Compute::ComputeNegDivergence()
{
  ComputeNegDivergence_kernel();
}
//==============================================================================
float Compute::ComputeTimestep()
{
  thrust::device_vector<float> tdv_velo;
  // enough space to hold each velocity component
  tdv_velo.resize((DataInfo->resolution[0]+1)*
		  (DataInfo->resolution[1]+1)*
		  (DataInfo->resolution[2]+1));
  float *velo_raw_ptr = thrust::raw_pointer_cast(tdv_velo.data());
  thrust::device_vector<float>::iterator iter;

  // TODO: do I have to copy the array?
  // TODO: write my own reduce algorithm

  CopyCudaArray(ca_VelocityX, Res[0], velo_raw_ptr);
  iter = thrust::max_element(tdv_velo.begin(), tdv_velo.end());
  float max_u = fabs(*iter);
  iter = thrust::min_element(tdv_velo.begin(), tdv_velo.end());
  float min_u = fabs(*iter);
  max_u = min_u > max_u ? min_u : max_u;

  CopyCudaArray(ca_VelocityY, Res[1], velo_raw_ptr);
  iter =thrust::max_element(tdv_velo.begin(), tdv_velo.end());
  float max_v = fabs(*iter);
  iter = thrust::min_element(tdv_velo.begin(), tdv_velo.end());
  float min_v = fabs(*iter);
  max_v = min_v > max_v ? min_v : max_v;

  CopyCudaArray(ca_VelocityZ, Res[2], velo_raw_ptr);  
  iter = thrust::max_element(tdv_velo.begin(), tdv_velo.end());
  float max_w = fabs(*iter);
  iter = thrust::min_element(tdv_velo.begin(), tdv_velo.end());
  float min_w = fabs(*iter);
  max_w = min_w > max_w ? min_w : max_w;

  float u_max = max(max(max_u,max_v),max_w);
  float dx = 
    min(min(DataInfo->spacing[0],DataInfo->spacing[1]),DataInfo->spacing[2]);
  float CFL = 0.25f;
  float dt;

  if (u_max > 0.0f) {
    dt = CFL*dx/u_max;
  }
  else {
    dt = 0.0f;
  }
  return dt;
}
//==============================================================================
void Compute::PressureUpdate()
{
  PressureUpdate_kernel();

  UpdateCudaArray(ca_VelocityX, Res[0], VelocityX);
  UpdateCudaArray(ca_VelocityY, Res[1], VelocityY);
  UpdateCudaArray(ca_VelocityZ, Res[2], VelocityZ);
}
// TODO: implement boundary conditions in textures
// TODO: modify rhs to account solid velocities
// TODO: build matrix A
